#include "hip/hip_runtime.h"
/**
	NVCSV : A CUDA-based CSV parser.
	File: nvcsv.cu
	Desc: Entry point for NVCSV.
	Author: Brandon Belna (bbelna)
**/

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <ctime>
#include "nvcsv.h"
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <string.h>
#include <limits.h>

int main(int argc, char** argv) {
	std::cout << "NVCSV Version " <<  NVCSV_VERSION << std::endl;
	if (argc == 1) {
		std::cout << "Usage: nvcsv [filename] [index count]" << std::endl;
		std::cout << "Currently built for CINF401." << std::endl;
		return 1;
	}
	int iC = atoi(*(argv+2));
	std::string fileName(*(argv+1)); 
	std::clock_t start1 = std::clock();
	FILE* f = fopen(fileName.c_str(), "r" );
	if (f == NULL) {
		std::cout << "failed to open " <<  fileName << ". Does file exist?" << std::endl;
		return 1;
	}
	std::cout << "Determining size of " << fileName << "..." << std::endl;
	fseek(f, 0, SEEK_END);
	struct stat st;
	stat(fileName.c_str(), &st);	
	long long fileSize = st.st_size; 
	thrust::device_vector<char> dev(fileSize); // the vector representing the
						   // file's data on the GPU's memory
	std::cout << "File size is " << fileSize << "." << std::endl;
	fclose(f);
	
	struct stat sb;
	char *p;
	int fd;

	fd = open (fileName.c_str(), O_RDONLY);
	if (fd == -1) {
		perror ("open");
		return 1;
	}

	if (fstat (fd, &sb) == -1) {
		perror ("fstat");
		return 1;
	}

	if (!S_ISREG (sb.st_mode)) {
		fprintf (stderr, "%s is not a file\n", "fileName");
		return 1;
	}

	p = (char*)mmap (0, fileSize, PROT_READ, MAP_SHARED, fd, 0);

	if (p == MAP_FAILED) {
		perror ("mmap");
		return 1;
	}

	if (close (fd) == -1) {
		perror ("close");
		return 1;
	}


	std::cout << "Copying file to GPU..." << std::endl;
	thrust::copy(p, p+fileSize, dev.begin());

	std::cout << "Counting lines..." << std::endl;
	long long cnt = thrust::count(dev.begin(), dev.end(), '\n'); // count the new lines in the file
	std::cout << "There are " << cnt << " total lines in the file." << std::endl;

	// find all new lines
	thrust::device_vector<int> devPos(cnt+1);
	devPos[0] = -1;
	
	std::cout << "Creating device_vector of newlines..." << std::endl;
	thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)fileSize),
		dev.begin(), devPos.begin()+1, is_break());
	
	std::cout << "Creating value arrays..." << std::endl;
	thrust::device_vector<char> vals(cnt*25); // where we'll store our values
	thrust::fill(vals.begin(), vals.end(), ' '); // pad whole vector with zeros

	thrust::device_vector<char*> dest(1);
	dest[0] = thrust::raw_pointer_cast(vals.data()); // destination pointer

	thrust::device_vector<unsigned int> index(1); 
	index[0] = 4; // we want the fifth column

	thrust::device_vector<unsigned int> destLen(1); 
	destLen[0] = 25; // max of 25 in length
	
	thrust::device_vector<unsigned int> indexCount(1);
	indexCount[0] = 1;

	thrust::device_vector<char> seperator(1);
	seperator[0] = ',';

	std::cout << "Parsing CSV file on GPU (this may take a while)..." << std::endl;
	thrust::counting_iterator<unsigned int> begin(0);
	parse_functor ff((const char*)thrust::raw_pointer_cast(dev.data()),(char**)thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(index.data()),
		thrust::raw_pointer_cast(indexCount.data()), thrust::raw_pointer_cast(seperator.data()), thrust::raw_pointer_cast(devPos.data()), thrust::raw_pointer_cast(destLen.data()));
	thrust::for_each(begin, begin+cnt, ff);

	thrust::device_vector<double> d_float(cnt);
	
	std::cout << "gpu_atof on wanted data..." << std::endl;
	indexCount[0] = iC;
	gpu_atof atof_ff((const char*)thrust::raw_pointer_cast(vals.data()),(double*)thrust::raw_pointer_cast(d_float.data()),
			thrust::raw_pointer_cast(indexCount.data()));
	thrust::for_each(begin, begin + cnt, atof_ff);

	std::cout.precision(10);
	for(int i = 0; i < 10; i++) {
		std::cout << d_float[i] << std::endl;
	}
	std::cout << "Success. Exiting..." << std::endl;
}
