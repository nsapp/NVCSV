#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <ctime>
#include "nvcsv.h"
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <string.h>

int main(int argc, char** argv) {
	std::cout << "NVCSV Version " <<  NVCSV_VERSION << std::endl;
	if (argc == 1) {
		std::cout << "Usage: nvcsv [filename]" << std::endl;
		std::cout << "As of now, currently only runs through file listed to see how long it takes to process." << std::endl;
		return 1;
	}
	std::string fileName(*(argv+1)); 
	std::clock_t start1 = std::clock();
	FILE* f = fopen(fileName.c_str(), "r" );
	if (f == NULL) {
		std::cout << "failed to open " <<  fileName << ". Does file exist?" << std::endl;
		return 1;
	}
	fseek(f, 0, SEEK_END);
	long fileSize = ftell(f);
	thrust::device_vector<char> dev(fileSize);
	fclose(f);
	
	struct stat sb;
	char *p;
	int fd;

	fd = open (fileName.c_str(), O_RDONLY);
	if (fd == -1) {
		perror ("open");
		return 1;
	}

	if (fstat (fd, &sb) == -1) {
		perror ("fstat");
		return 1;
	}

	if (!S_ISREG (sb.st_mode)) {
		fprintf (stderr, "%s is not a file\n", "fileName");
		return 1;
	}

	p = (char*)mmap (0, fileSize, PROT_READ, MAP_SHARED, fd, 0);

	if (p == MAP_FAILED) {
		perror ("mmap");
		return 1;
	}

	if (close (fd) == -1) {
		perror ("close");
		return 1;
	}

	thrust::copy(p, p+fileSize, dev.begin());

	int cnt = thrust::count(dev.begin(), dev.end(), '\n');
	std::cout << "There are " << cnt << " total lines in a file" << std::endl;

	thrust::device_vector<int> dev_pos(cnt+1);
	dev_pos[0] = -1;

	thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)fileSize),
					dev.begin(), dev_pos.begin()+1, is_break());

	thrust::device_vector<char> res(cnt*20);
	thrust::fill(res.begin(), res.end(), 0);

	thrust::device_vector<char*> dest(1);
	dest[0] = thrust::raw_pointer_cast(res.data());

	thrust::device_vector<unsigned int> ind(1); //fields positions
	ind[0] = 5;

	thrust::device_vector<unsigned int> dest_len(1); //fields max lengths
	dest_len[0] = 20;

	thrust::device_vector<unsigned int> ind_cnt(1); //fields count
	ind_cnt[0] = 10;

	thrust::device_vector<char> sep(1);
	sep[0] = ',';

	thrust::counting_iterator<unsigned int> begin(0);
	parse_functor ff((const char*)thrust::raw_pointer_cast(dev.data()),(char**)thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(ind.data()),
					 thrust::raw_pointer_cast(ind_cnt.data()), thrust::raw_pointer_cast(sep.data()), thrust::raw_pointer_cast(dev_pos.data()), thrust::raw_pointer_cast(dest_len.data()));
	thrust::for_each(begin, begin + cnt, ff); // now dev_pos vector contains the indexes of new line characters

	std::cout<< "time0 " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << '\n';
	
	thrust::device_vector<double> d_float(cnt);

	gpu_atof atof_ff((const char*)thrust::raw_pointer_cast(res.data()),(double*)thrust::raw_pointer_cast(d_float.data()),
					 thrust::raw_pointer_cast(ind_cnt.data()));
	thrust::for_each(begin, begin + cnt, atof_ff);

	std::cout.precision(10);
	for(int i = 0; i < 10; i++)
		std::cout << d_float[i] << std::endl;

	return 0;

}
