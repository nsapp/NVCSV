#include "hip/hip_runtime.h"
/**
	NVCSV : A CUDA-based CSV parser.
	File: nvcsv.cu
	Desc: Entry point for NVCSV.
	Author: Brandon Belna (bbelna)
**/

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <ctime>
#include "nvcsv.h"
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <string.h>
#include <limits.h>

#define OFFSET 100000	// offset for when the file has near max int lines

int parseCSV(std::string, int, int);	// foreword dec of the parseCSV func

// simple message function
void msg(std::string m) {
	std::cout << m << std::endl;
}

int main(int argc, char** argv) {
	// TODO: replace most std::cout statements with msg()
	std::cout << "NVCSV Version " <<  NVCSV_VERSION << std::endl;
	msg("Initializing CUDA context...");
	hipFree(0);	// this is a shorthand which initializes the CUDA context
		    	// normally the CUDA context will initialize on the first CUDA command
	if (argc < 4) { // if our arguments are insufficient
		msg("Currently only supports grabbing data from a column of a CSV file.");
		msg("Usage: nvcsv [filename] [index] [field max length]");
		return 1;
	}
	/*
		TODO: allow [filename] to be replaced w/ a directory and have NVCSV parse the whole directory.
		You i.e. would loop parseCSV over the whole directory.

		There's a way (using stat I believe) to be able to tell if a string is a file or a directory.
	*/
	int fieldMaxLength = atoi(*(argv+3));	// field max length
	int index = atoi(*(argv+2));	// the column to grab
	if (index < 0) {
		msg("Error: index must be > 0. Aborting...");
		return -1;
	}
	if (fieldMaxLength <= 0) {
		msg("Error: max length must be > 0. Aborting...");
		return -1;
	}
	std::string fileName(*(argv+1));
	parseCSV(fileName, fieldMaxLength, index);	// parse the csv
	exit(0);	// return
}

int parseCSV(std::string _fileName, int _maxLength, int _parseIndex) {
	// variable init
	int maxLength = _maxLength;
	int parseIndex = _parseIndex;
	std::string fileName = _fileName;
	std::clock_t start1 = std::clock();

	// open file, determine size
	FILE* f = fopen(fileName.c_str(), "r" );
	std::cout << "Determining size of " << fileName << "..." << std::endl;
	fseek(f, 0, SEEK_END);	// seak towards end of file
	struct stat st;
	stat(fileName.c_str(), &st);	// get the file size
	long long fileSize = st.st_size;	// give that size a name
	std::cout << "File size is " << fileSize << "." << std::endl;
	fclose(f); // close the file

	struct stat sb;	// get another statr
	char *p;	// the pointer to our file data
	int fd;		// integer representing the file

	fd = open (fileName.c_str(), O_RDONLY);	// open

	// error handling
	if (fd == -1) {
		perror ("open");
		return 1;
	}

	if (fstat (fd, &sb) == -1) {
		perror ("fstat");
		return 1;
	}

	if (!S_ISREG (sb.st_mode)) {
		fprintf (stderr, "%s is not a file\n", "fileName");
		return 1;
	}	
	
	// create a shared memory map
	p = (char*)mmap (0, fileSize, PROT_READ, MAP_SHARED, fd, 0);

	if (p == MAP_FAILED) {
		perror ("mmap");
		return 1;
	}

	if (close (fd) == -1) {
		perror ("close");
		return 1;
	}
	
	
	/*
		TODO: if fileSize >= INT_MAX-OFFSET, we should set our fileSize variable
		to be that of INT_MAX-OFFSET, and then jump backwards until we reach a newline
		character. This code doesn't work in its current state, I believe.

		After you parse INT_MAX-OFFSET, you need to parse the next INT_MAX-OFFSET (or whatever's
		left of the file). Obviously, there's going to be a problem if you try to get the GPU
		to process the INT_MAX-OFFEST'th byte because the problem lies in the fact that throwing
		such a number into thrust causes the program to crash.

		Thus, you need to create a pointer whose first entry is that of INT_MAX-OFFSET; so that when
		the GPU references the INT_MAX-OFFEST'th byte it's really accessing the first entry of that pointer.
		This removes the problem that thrust can't directly handle files whose size are near INT_MAX.
	*/
	if (fileSize >= INT_MAX-OFFSET) {
		msg("File size >= INT_MAX. Splitting CSV file...");
		// jump to INT MAX. go backwards until new line.
		FILE* fd2 = fopen(fileName.c_str(), "r" );	
		fileSize = INT_MAX-OFFSET;
		fseek(fd2, fileSize, SEEK_SET);
		char t = fgetc(fd2);
		while (t != '\n') {
			fileSize--;
			fseek(fd2, fileSize, SEEK_SET);
			t = fgetc(fd2);
		}	
		fclose(fd2);
	}

	// GPU parsing code
	std::cout << "New file size: " << fileSize << std::endl;
	thrust::device_vector<char> dev(fileSize); // the vector representing the
						   // file's data on the GPU's memory
	msg("Copying file to GPU (this may take a while)...");
	thrust::copy(p, p+fileSize, dev.begin());
	msg("Successful copy to GPU.");
	msg("Counting lines...");
	thrust::device_vector<unsigned long long int> cnt(1);
	cnt[0] = thrust::count(thrust::device, dev.begin(), dev.end(), '\n'); // count the new lines in the file
	std::cout << "There are " << cnt[0] << " total lines in the file." << std::endl;


	/*
		This is ALL GPU processing code. For the most part you shouldn't have
		to touch this.
	*/
	// find all new lines
	thrust::device_vector<int> devPos(cnt[0]+1);
	devPos[0] = -1;
	
	msg("Creating device_vector of newlines...");
	thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)fileSize),
		dev.begin(), devPos.begin()+1, is_break());
	
	msg("Creating value arrays...");
	thrust::device_vector<char> vals(cnt[0]*25); // where we'll store our values
	thrust::fill(vals.begin(), vals.end(), ' '); // pad whole vector with zeros

	msg("Establishing destination pointer...");
	thrust::device_vector<char*> dest(1);
	dest[0] = thrust::raw_pointer_cast(vals.data()); // destination pointer

	msg("Establishing index vector...");
	thrust::device_vector<unsigned int> index(1); 
	index[0] = parseIndex;

	msg("Establishing max length of field...");
	thrust::device_vector<unsigned int> destLen(1); 
	destLen[0] = maxLength;
	
	thrust::device_vector<unsigned int> indexCount(1);
	indexCount[0] = 1;

	thrust::device_vector<char> seperator(1);
	seperator[0] = ',';

	msg("Parsing column...");
	thrust::counting_iterator<unsigned int> begin(0);
	parse_functor ff((const char*)thrust::raw_pointer_cast(dev.data()),(char**)thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(index.data()),
		thrust::raw_pointer_cast(indexCount.data()), thrust::raw_pointer_cast(seperator.data()), thrust::raw_pointer_cast(devPos.data()), thrust::raw_pointer_cast(destLen.data()));
	thrust::for_each(begin, begin + cnt[0], ff);
	msg("Successful parse.");
	thrust::device_vector<double> d_float(cnt[0]);
	
	msg("gpu_atof on wanted data...");
	indexCount[0] = maxLength;
	gpu_atof atof_ff((const char*)thrust::raw_pointer_cast(vals.data()),(double*)thrust::raw_pointer_cast(d_float.data()),
			thrust::raw_pointer_cast(indexCount.data()));
	thrust::for_each(begin, begin + cnt[0], atof_ff);
	msg("Successful gpu_atof.");

	msg("Here are the first 10 entries of your desired column:");
	std::cout.precision(10);
	for(int i = 0; i < 10; i++) {
		std::cout << d_float[i] << std::endl;
	}
	msg("Cleaning...");
}
