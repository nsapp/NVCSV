#include "hip/hip_runtime.h"
/**
	NVCSV : A CUDA-based CSV parser.
	File: nvcsv.cu
	Desc: Entry point for NVCSV.
	Author: Brandon Belna (bbelna)
**/

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <ctime>
#include "nvcsv.h"
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <string.h>
#include <limits.h>

void msg(std::string m) {
	std::cout << m << std::endl;
}

int main(int argc, char** argv) {
	// TODO: replace most std::cout statements with msg()
	std::cout << "NVCSV Version " <<  NVCSV_VERSION << std::endl;
	if (argc < 4) {
		std::cout << "Currently only supports grabbing data from a column of a CSV file." << std::endl;
		std::cout << "Usage: nvcsv [filename] [index] [field max length]" << std::endl;
		return 1;
	}
	int iC = atoi(*(argv+3));
	int dI = atoi(*(argv+2));
	if (dI < 0) {
		msg("Error: index must be > 0. Aborting...");
		return -1;
	}
	if (iC <= 0) {
		msg("Error: max length must be > 0. Aborting...");
		return -1;
	}
	std::string fileName(*(argv+1)); 
	std::clock_t start1 = std::clock();
	FILE* f = fopen(fileName.c_str(), "r" );
	if (f == NULL) {
		std::cout << "Error: Failed to open " <<  fileName << ". Does file exist? Aborting..." << std::endl;
		return -1;
	}
	std::cout << "Determining size of " << fileName << "..." << std::endl;
	fseek(f, 0, SEEK_END);
	struct stat st;
	stat(fileName.c_str(), &st);	
	long long fileSize = st.st_size; 
	thrust::device_vector<char> dev(fileSize); // the vector representing the
						   // file's data on the GPU's memory
	std::cout << "File size is " << fileSize << "." << std::endl;
	fclose(f);
	
	struct stat sb;
	char *p;
	int fd;

	fd = open (fileName.c_str(), O_RDONLY);
	if (fd == -1) {
		perror ("open");
		return 1;
	}

	if (fstat (fd, &sb) == -1) {
		perror ("fstat");
		return 1;
	}

	if (!S_ISREG (sb.st_mode)) {
		fprintf (stderr, "%s is not a file\n", "fileName");
		return 1;
	}

	p = (char*)mmap (0, fileSize, PROT_READ, MAP_SHARED, fd, 0);

	if (p == MAP_FAILED) {
		perror ("mmap");
		return 1;
	}

	if (close (fd) == -1) {
		perror ("close");
		return 1;
	}


	std::cout << "Copying file to GPU (this may take a while)..." << std::endl;
	thrust::copy(p, p+fileSize, dev.begin());
	msg("Successful copy to GPU.");
	std::cout << "Counting lines..." << std::endl;
	long long cnt = thrust::count(dev.begin(), dev.end(), '\n'); // count the new lines in the file
	std::cout << "There are " << cnt << " total lines in the file." << std::endl;

	// find all new lines
	thrust::device_vector<int> devPos(cnt+1);
	devPos[0] = -1;
	
	std::cout << "Creating device_vector of newlines..." << std::endl;
	thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)fileSize),
		dev.begin(), devPos.begin()+1, is_break());
	
	std::cout << "Creating value arrays..." << std::endl;
	thrust::device_vector<char> vals(cnt*25); // where we'll store our values
	thrust::fill(vals.begin(), vals.end(), ' '); // pad whole vector with zeros

	msg("Establushing destination pointer...");
	thrust::device_vector<char*> dest(1);
	dest[0] = thrust::raw_pointer_cast(vals.data()); // destination pointer

	msg("Establishing index vector...");
	thrust::device_vector<unsigned int> index(1); 
	index[0] = dI;

	msg("Establishing max length of field...");
	thrust::device_vector<unsigned int> destLen(1); 
	destLen[0] = iC;
	
	thrust::device_vector<unsigned int> indexCount(1);
	indexCount[0] = 1;

	thrust::device_vector<char> seperator(1);
	seperator[0] = ',';

	std::cout << "Parsing column..." << std::endl;
	thrust::counting_iterator<unsigned int> begin(0);
	parse_functor ff((const char*)thrust::raw_pointer_cast(dev.data()),(char**)thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(index.data()),
		thrust::raw_pointer_cast(indexCount.data()), thrust::raw_pointer_cast(seperator.data()), thrust::raw_pointer_cast(devPos.data()), thrust::raw_pointer_cast(destLen.data()));
	thrust::for_each(begin, begin+cnt, ff);
	msg("Successful parse.");
	thrust::device_vector<double> d_float(cnt);
	
	std::cout << "gpu_atof on wanted data..." << std::endl;
	indexCount[0] = iC;
	gpu_atof atof_ff((const char*)thrust::raw_pointer_cast(vals.data()),(double*)thrust::raw_pointer_cast(d_float.data()),
			thrust::raw_pointer_cast(indexCount.data()));
	thrust::for_each(begin, begin + cnt, atof_ff);
	msg("Successful gpu_atof.");

	msg("Here are the first 10 entries of your desired column:");
	std::cout.precision(10);
	for(int i = 0; i < 10; i++) {
		std::cout << d_float[i] << std::endl;
	}
	msg("Cleaning...");
	// TODO: clean tasks?
	std::cout << "Terminating NVCSV..." << std::endl;
}
